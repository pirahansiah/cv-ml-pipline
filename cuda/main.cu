#include <iostream>
#include <string>
#include <hip/hip_runtime.h>


// Simple integer to string conversion that works with CUDA
std::string intToString(int value) {
    if (value == 0) return "0";
    
    std::string result;
    bool negative = value < 0;
    if (negative) value = -value;
    
    while (value > 0) {
        result = char('0' + value % 10) + result;
        value /= 10;
    }
    
    if (negative) result = "-" + result;
    return result;
}

static std::string decodeCudaVersion(int v) {
    int major = v / 1000;
    int minor = (v % 1000) / 10;
    return intToString(major) + "." + intToString(minor);
}

int main() {
    int driverVersion = 0, runtimeVersion = 0;
    hipError_t rc1 = hipDriverGetVersion(&driverVersion);
    hipError_t rc2 = hipRuntimeGetVersion(&runtimeVersion);

    std::cout << "CUDA driver rc=" << (int)rc1
              << " version=" << driverVersion
              << " (" << decodeCudaVersion(driverVersion) << ")\n";

    std::cout << "CUDA runtime rc=" << (int)rc2
              << " version=" << runtimeVersion
              << " (" << decodeCudaVersion(runtimeVersion) << ")\n";

    return 0;
}