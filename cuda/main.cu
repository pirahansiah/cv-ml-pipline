
#include <iostream>
#include <hip/hip_runtime.h>

//#include <opencv2/core.hpp>

static std::string decodeCudaVersion(int v) {
    int major = v / 1000;
    int minor = (v % 1000) / 10;
    return std::to_string(major) + "." + std::to_string(minor);
}

int main() {
    int driverVersion = 0, runtimeVersion = 0;
    hipError_t rc1 = hipDriverGetVersion(&driverVersion);
    hipError_t rc2 = hipRuntimeGetVersion(&runtimeVersion);

    std::cout << "CUDA driver rc=" << (int)rc1
              << " version=" << driverVersion
              << " (" << decodeCudaVersion(driverVersion) << ")\n";

    std::cout << "CUDA runtime rc=" << (int)rc2
              << " version=" << runtimeVersion
              << " (" << decodeCudaVersion(runtimeVersion) << ")\n";

    // std::cout << "OpenCV version: " << CV_VERSION
    //           << " (cv::getVersionString=" << cv::getVersionString() << ")\n";
    return 0;
}